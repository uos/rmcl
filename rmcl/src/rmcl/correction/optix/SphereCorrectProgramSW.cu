#include "hip/hip_runtime.h"
#include <optix.h>
#include "rmcl/correction/optix/CorrectionDataOptix.hpp"
#include <rmagine/math/types.h>
#include <rmagine/map/optix/optix_sbt.h>

namespace rm = rmagine;

extern "C" {
__constant__ rmcl::SphereCorrectionDataSW mem;
}

extern "C" __global__ void __raygen__rg()
{
    const float dist_thresh = mem.params->max_distance;

    // Lookup our location within the launch grid
    const uint3 idx = optixGetLaunchIndex();
    const uint3 dim = optixGetLaunchDimensions();

    const unsigned int Nvertical = mem.model->getHeight();
    const unsigned int Nhorizontal = mem.model->getWidth();

    const unsigned int pid = idx.z * dim.y * dim.x + idx.y * dim.x + idx.x;

    if(pid < mem.Nposes)
    {
        const float rangeMin = mem.model->range.min;
        const float rangeMax = mem.model->range.max;
        
        const rm::Transform Tsb = mem.Tsb[0];
        const rm::Transform Tbm = mem.Tbm[pid];
        const rm::Transform Tsm = Tbm * Tsb;
        const rm::Transform Tms = Tsm.inv();

        // TODO: is it possible to not doing optixTrace twice?
        // - solution: fixing the rotation center to the robot's base
        // -- expected cons: slower convergence
        // -- expected pros:
        // --- better stability. never glitch through walls: better recovery after an error
        // --- faster: 2x
        // Solution was not good, correspondences need sometime force the problem
        // to have another rotation center. For example: Depth Cameras having 
        // having correspondences mostly in front of the sensor, thus the rotation must be 
        // around enother center, not the robots.
        // But how to weighted fuse measurements then?

        rm::Vector Dmean = {0.0, 0.0, 0.0};
        rm::Vector Mmean = {0.0, 0.0, 0.0};
        unsigned int Ncorr = 0;
        rm::Matrix3x3 C;
        C.setZeros();
        
        // Computing Means
        for(unsigned int vid = 0; vid < Nvertical; vid++)
        {
            for(unsigned int hid = 0; hid < Nhorizontal; hid++)
            {
                // ids of results
                const unsigned int loc_id = mem.model->getBufferId(vid, hid);
                
                const float real_range = mem.ranges[loc_id];
                if(real_range < rangeMin || real_range > rangeMax)
                {
                    continue;
                }

                const rm::Vector ray_dir_s = mem.model->getDirection(vid, hid);
                const rm::Vector ray_dir_m = Tsm.R * ray_dir_s;

                unsigned int p0, p1, p2, p3;
                optixTrace(
                        mem.handle,
                        make_float3(Tsm.t.x, Tsm.t.y, Tsm.t.z),
                        make_float3(ray_dir_m.x, ray_dir_m.y, ray_dir_m.z),
                        0.0f,               // Min intersection distance
                        rangeMax,                   // Max intersection distance
                        0.0f,                       // rayTime -- used for motion blur
                        OptixVisibilityMask( 1 ),   // Specify always visible
                        OPTIX_RAY_FLAG_DISABLE_ANYHIT,
                        0,          // SBT offset
                        1,             // SBT stride
                        0,          // missSBTIndex
                        p0, p1, p2, p3);

                const float range = __uint_as_float( p0 );
                if(range > rangeMax)
                {
                    continue;
                }

                rm::Vector nint_m = {
                    __uint_as_float( p1 ),
                    __uint_as_float( p2 ),
                    __uint_as_float( p3 )
                };

                nint_m.normalizeInplace();

                // going to sensor space
                const rm::Vector preal_s = ray_dir_s * real_range;
                const rm::Vector pint_s = ray_dir_s * range;

                rm::Vector nint_s = Tms.R * nint_m;
                
                // if(nint_s.dot(ray_dir_s) > 0.0)
                // {
                //     nint_s *= -1.0;
                // }

                const float signed_plane_dist = (pint_s - preal_s).dot(nint_s);
                const rm::Vector pmesh_s = preal_s + nint_s * signed_plane_dist;
                const float dist_sqrt = (pmesh_s - preal_s).l2normSquared();

                if(dist_sqrt < dist_thresh * dist_thresh)
                {
                    const rm::Vector preal_b = Tsb * preal_s;
                    const rm::Vector pmesh_b = Tsb * pmesh_s;
                    // Online update: Covariance and means 
                    // - wrong: https://en.wikipedia.org/wiki/Algorithms_for_calculating_variance 
                    // use the following equations instead
                    {
                        const float N_1 = static_cast<float>(Ncorr);
                        const float N = static_cast<float>(Ncorr + 1);
                        const float w1 = N_1/N;
                        const float w2 = 1.0/N;

                        const rm::Vector d_mean_old = Dmean;
                        const rm::Vector m_mean_old = Mmean;

                        const rm::Vector d_mean_new = d_mean_old * w1 + preal_b * w2; 
                        const rm::Vector m_mean_new = m_mean_old * w1 + pmesh_b * w2;

                        auto P1 = (pmesh_b - m_mean_new).multT(preal_b - d_mean_new);
                        auto P2 = (m_mean_old - m_mean_new).multT(d_mean_old - d_mean_new);

                        // write
                        Dmean = d_mean_new;
                        Mmean = m_mean_new;
                        C = C * w1 + P1 * w2 + P2 * w1;
                        Ncorr = Ncorr + 1;
                    }
                }
            }
        }

        mem.Ncorr[pid] = Ncorr;
        mem.C[pid] = C;
        mem.m1[pid] = Dmean;
        mem.m2[pid] = Mmean;
    }
}

extern "C" __global__ void __miss__ms()
{
    optixSetPayload_0( __float_as_uint( mem.model->range.max + 1.0f ) );
}

extern "C" __global__ void __closesthit__ch()
{
    const float t = optixGetRayTmax();
    const unsigned int face_id = optixGetPrimitiveIndex();
    const unsigned int inst_id = optixGetInstanceId();
    const unsigned int gas_id = optixGetSbtGASIndex();

    rm::OptixSceneSBT* scene_data  = reinterpret_cast<rm::OptixSceneSBT*>( optixGetSbtDataPointer() );

    rm::OptixMeshSBT* mesh_data = nullptr;
    if(scene_data->type == rm::OptixSceneType::INSTANCES)
    {
        // instance hierarchy
        rm::OptixSceneSBT* inst_scene = scene_data->geometries[inst_id].inst_data.scene;
        mesh_data = &(inst_scene->geometries[gas_id].mesh_data);
    } else {
        mesh_data = &scene_data->geometries[gas_id].mesh_data;
    }

    const float3 normal = make_float3(
        mesh_data->face_normals[face_id].x, 
        mesh_data->face_normals[face_id].y, 
        mesh_data->face_normals[face_id].z);

    float3 normal_world = optixTransformNormalFromObjectToWorldSpace(normal);

    optixSetPayload_0( __float_as_uint( t ) );
    optixSetPayload_1( __float_as_uint( normal_world.x ) );
    optixSetPayload_2( __float_as_uint( normal_world.y ) );
    optixSetPayload_3( __float_as_uint( normal_world.z ) );
}