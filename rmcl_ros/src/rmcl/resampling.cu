#include "hip/hip_runtime.h"
#include "rmcl_ros/rmcl/resampling.cuh"
#include <hip/hip_runtime.h>

#include <iostream>

#include <hiprand.h>

namespace rm = rmagine;

namespace rmcl
{

__global__ 
void init_curand_kernel(hiprandState* states, const unsigned int N)
{
  unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;

  if(idx < N)
  {
    hiprand_init(1234, idx, 0, &states[idx]);
  }
}

void init_curand(rmagine::MemoryView<hiprandState, rmagine::VRAM_CUDA>& curand_states)
{
  constexpr unsigned int blockSize = 1024;
  const unsigned int gridSize = (curand_states.size() + blockSize - 1) / blockSize;

  init_curand_kernel<<<gridSize, blockSize>>>(curand_states.raw(), curand_states.size());
}

__device__
SimpleLikelihoodStats merge(const SimpleLikelihoodStats s1, const SimpleLikelihoodStats s2)
{
  SimpleLikelihoodStats sm;
  sm.sum = s1.sum + s2.sum;
  sm.max = max(s1.max, s2.max);
  return sm;
}

template<unsigned int blockSize>
__global__ void simple_stats_kernel(
    const rm::Transform* poses,
    const ParticleAttributes* attrs,
    const unsigned int N,
    SimpleLikelihoodStats* res)
{
  __shared__ SimpleLikelihoodStats sdata[blockSize];
  
  const unsigned int tid = threadIdx.x;
  const unsigned int globId = N * blockIdx.x + threadIdx.x;
  const unsigned int rows = (N + blockSize - 1) / blockSize;

  sdata[tid].sum = 0.0; // TODO: this is a trick, but not good
  sdata[tid].max = 0.0;

  for(unsigned int i=0; i<rows; i++)
  {
    if(globId + blockSize * i < N)
    {
      const float L = attrs[globId + blockSize * i].likelihood.mean;
      sdata[threadIdx.x].sum += L;
      sdata[tid].max = max(sdata[tid].max, L);
    }
  }
  __syncthreads();

  for(unsigned int s = blockSize / 2; s > 0; s >>= 1)
  {
    if(tid < s)
    {
      sdata[tid] = merge(sdata[tid], sdata[tid + s]);
    }
    __syncthreads();
  }

  if(tid == 0)
  {
    res[blockIdx.x] = sdata[0];
  }
}

void compute_stats(
  const rm::MemoryView<rm::Transform, rmagine::VRAM_CUDA>& particle_poses,
  const rm::MemoryView<ParticleAttributes, rmagine::VRAM_CUDA>& particle_attrs,
  rm::MemoryView<SimpleLikelihoodStats, rmagine::VRAM_CUDA> stats)
{
  const unsigned int n_outputs = stats.size(); // also number of blocks
  constexpr unsigned int n_threads = 512; // also shared mem

  simple_stats_kernel<n_threads> <<<n_outputs, n_threads>>>(particle_poses.raw(), particle_attrs.raw(), particle_poses.size(), stats.raw());
}


__device__ unsigned int lcg_rand(unsigned int &state) {
  // Constants from Numerical Recipes
  const unsigned int A = 1664525;
  const unsigned int C = 1013904223;
  state = A * state + C;
  return state;
}

__device__ float lcg_rand_flt(unsigned int& state) {
  return lcg_rand(state) / (float)UINT_MAX;
}


__global__
void residual_resample_kernel(
  const rmagine::Transform* particle_poses,
  const ParticleAttributes* particle_attrs,
  const SimpleLikelihoodStats* stats,
  hiprandState* rstates,
  rmagine::Transform* particle_poses_new,
  ParticleAttributes* particle_attrs_new,
  const unsigned int n_particles,
  const ParticleUpdateDynamicConfig config)
{
  const unsigned int pid = blockIdx.x * blockDim.x + threadIdx.x;
  

  const float min_noise_tx = 0.03;
  const float min_noise_ty = 0.03;
  const float min_noise_tz = 0.0;
  
  const float min_noise_roll  = 0.0;
  const float min_noise_pitch = 0.0;
  const float min_noise_yaw   = 0.01;

  const float likelihood_forget_per_meter = 0.3;
  const float likelihood_forget_per_radian = 0.2;

  

  if(pid < n_particles)
  {
    hiprandState& rstate = rstates[pid];



    const float L_max = stats->max;
    const float L_sum = stats->sum;

    const rm::Transform pose = particle_poses[pid];
    const ParticleAttributes attrs = particle_attrs[pid];
    
    // init state with randomness from actual PRNG (CPU).


    // unsigned int state = *reinterpret_cast<const unsigned int*>(&pose.t.x);
    const unsigned int rand_int = hiprand(&rstate);

    // hiprand_discrete()

    const unsigned int rand_id = rand_int % n_particles;
    const float rand_flt = (rand_int / (float)UINT_MAX); // betwen 0 and 1

    const float L = attrs.likelihood.mean;
    const float L_sum_normed = L / L_sum;
    const float L_max_normed = L / L_max;

    rm::Transform pose_new;
    ParticleAttributes attrs_new;

    if(rand_flt < L_max_normed)
    {
      // keep this particle
      pose_new = pose;
      attrs_new = attrs;
    } else {
      // otherwise take a random other particle
      pose_new = particle_poses[rand_id];
      attrs_new = particle_attrs[rand_id];
    }

    // float noise_tx = min_noise_tx / L_max_normed;
    // float noise_ty = min_noise_ty / L_max_normed;
    // float noise_tz = min_noise_tz / L_max_normed;

    // float noise_roll  = min_noise_roll  / L_max_normed;
    // float noise_pitch = min_noise_pitch / L_max_normed;
    // float noise_yaw   = min_noise_yaw   / L_max_normed;

    // pose_new.t.x += Nd(*rand_gen_) * noise_tx;
    // pose_new.t.y += Nd(*rand_gen_) * noise_ty;
    // pose_new.t.z += Nd(*rand_gen_) * noise_tz;


    particle_poses_new[pid] = pose_new;
    particle_attrs_new[pid] = attrs_new;
  }
}

void residual_resample(
  const rmagine::MemoryView<rmagine::Transform, rmagine::VRAM_CUDA> particle_poses,
  const rmagine::MemoryView<ParticleAttributes, rmagine::VRAM_CUDA> particle_attrs,
  const rmagine::MemoryView<SimpleLikelihoodStats, rmagine::VRAM_CUDA> stats,
  rmagine::MemoryView<hiprandState, rmagine::VRAM_CUDA> rstates,
  rmagine::MemoryView<rmagine::Transform, rmagine::VRAM_CUDA> particle_poses_new,
  rmagine::MemoryView<ParticleAttributes, rmagine::VRAM_CUDA> particle_attrs_new,
  const ParticleUpdateDynamicConfig& config)
{
  const unsigned int n_particles = particle_poses.size();
  constexpr unsigned int blockSize = 1024;
  const unsigned int gridSize = (particle_poses.size() + blockSize - 1) / blockSize;

  residual_resample_kernel<<<gridSize, blockSize>>>(
    particle_poses.raw(), particle_attrs.raw(), stats.raw(), rstates.raw(),
    particle_poses_new.raw(), particle_attrs_new.raw(), n_particles, config);
}




} // namespace rmcl