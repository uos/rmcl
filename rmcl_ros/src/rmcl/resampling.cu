#include "hip/hip_runtime.h"
#include "rmcl_ros/rmcl/resampling.cuh"
#include <hip/hip_runtime.h>

#include <iostream>

#include <hiprand.h>

namespace rm = rmagine;

namespace rmcl
{

__global__ 
void init_curand_kernel(hiprandState* states, const unsigned int N)
{
  unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;

  if(idx < N)
  {
    hiprand_init(1234, idx, 0, &states[idx]);
  }
}

void init_curand(rmagine::MemoryView<hiprandState, rmagine::VRAM_CUDA>& curand_states)
{
  constexpr unsigned int blockSize = 1024;
  const unsigned int gridSize = (curand_states.size() + blockSize - 1) / blockSize;

  init_curand_kernel<<<gridSize, blockSize>>>(curand_states.raw(), curand_states.size());
}

__device__
SimpleLikelihoodStats merge(const SimpleLikelihoodStats s1, const SimpleLikelihoodStats s2)
{
  SimpleLikelihoodStats sm;
  sm.sum = s1.sum + s2.sum;
  sm.max = max(s1.max, s2.max);
  return sm;
}

template<unsigned int blockSize>
__global__ void simple_stats_kernel(
    const rm::Transform* poses,
    const ParticleAttributes* attrs,
    const unsigned int N,
    SimpleLikelihoodStats* res)
{
  __shared__ SimpleLikelihoodStats sdata[blockSize];
  
  const unsigned int tid = threadIdx.x;
  const unsigned int globId = N * blockIdx.x + threadIdx.x;
  const unsigned int rows = (N + blockSize - 1) / blockSize;

  sdata[tid].sum = 0.0; // TODO: this is a trick, but not good
  sdata[tid].max = 0.0;

  for(unsigned int i=0; i<rows; i++)
  {
    if(globId + blockSize * i < N)
    {
      const float L = attrs[globId + blockSize * i].likelihood.mean;
      sdata[threadIdx.x].sum += L;
      sdata[tid].max = max(sdata[tid].max, L);
    }
  }
  __syncthreads();

  for(unsigned int s = blockSize / 2; s > 0; s >>= 1)
  {
    if(tid < s)
    {
      sdata[tid] = merge(sdata[tid], sdata[tid + s]);
    }
    __syncthreads();
  }

  if(tid == 0)
  {
    res[blockIdx.x] = sdata[0];
  }
}

void compute_stats(
  const rm::MemoryView<rm::Transform, rmagine::VRAM_CUDA>& particle_poses,
  const rm::MemoryView<ParticleAttributes, rmagine::VRAM_CUDA>& particle_attrs,
  rm::MemoryView<SimpleLikelihoodStats, rmagine::VRAM_CUDA> stats)
{
  const unsigned int n_outputs = stats.size(); // also number of blocks
  constexpr unsigned int n_threads = 512; // also shared mem

  simple_stats_kernel<n_threads> <<<n_outputs, n_threads>>>(particle_poses.raw(), particle_attrs.raw(), particle_poses.size(), stats.raw());
}


__device__ unsigned int lcg_rand(unsigned int &state) {
  // Constants from Numerical Recipes
  const unsigned int A = 1664525;
  const unsigned int C = 1013904223;
  state = A * state + C;
  return state;
}

__device__ float lcg_rand_flt(unsigned int& state) {
  return lcg_rand(state) / (float)UINT_MAX;
}


__global__
void residual_resample_kernel(
  const rmagine::Transform* particle_poses,
  const ParticleAttributes* particle_attrs,
  const SimpleLikelihoodStats* stats,
  hiprandState* rstates,
  rmagine::Transform* particle_poses_new,
  ParticleAttributes* particle_attrs_new,
  const unsigned int n_particles,
  const ParticleUpdateDynamicConfig config)
{
  const unsigned int pid = blockIdx.x * blockDim.x + threadIdx.x;

  const float min_noise_tx = 0.03;
  const float min_noise_ty = 0.03;
  const float min_noise_tz = 0.0;
  
  const float min_noise_roll  = 0.0;
  const float min_noise_pitch = 0.0;
  const float min_noise_yaw   = 0.01;

  const float likelihood_forget_per_meter = 0.3;
  const float likelihood_forget_per_radian = 0.2;

  const float L_max = stats->max;
  const float L_sum = stats->sum;

  if(pid < n_particles)
  {
    // Get random number
    hiprandState& rstate = rstates[pid];
    const unsigned int random_int = hiprand(&rstate);
    const unsigned int random_index = random_int % n_particles;

    // sample around this pose
    const rm::Transform pose = particle_poses[random_index];
    const ParticleAttributes attrs = particle_attrs[random_index];

    const float L = attrs.likelihood.mean;
    const float L_sum_normed = L / L_sum; // all L_normed are in sum 1; in [0, 1]
    const float L_max_normed = L / L_max; // L / L_max = L_normed2; in [0, 1]
    
    const size_t n_expected_insertions = L_sum_normed * n_particles;
    // const size_t n_insertions_left = n_particles - insertion_idx;

    // const size_t n_insertions = 
    //   (n_expected_insertions <= n_insertions_left) ? 
    //   n_expected_insertions : n_insertions_left;

    // // sample from this pose
    // const rm::Transform pose =       particle_poses[random_index];
    // const ParticleAttributes attrs = particle_attrs[random_index];
    
    // particle_poses_new[pid] = pose_new;
    // particle_attrs_new[pid] = attrs_new;
  }
}

void residual_resample(
  const rmagine::MemoryView<rmagine::Transform, rmagine::VRAM_CUDA> particle_poses,
  const rmagine::MemoryView<ParticleAttributes, rmagine::VRAM_CUDA> particle_attrs,
  const rmagine::MemoryView<SimpleLikelihoodStats, rmagine::VRAM_CUDA> stats,
  rmagine::MemoryView<hiprandState, rmagine::VRAM_CUDA> rstates,
  rmagine::MemoryView<rmagine::Transform, rmagine::VRAM_CUDA> particle_poses_new,
  rmagine::MemoryView<ParticleAttributes, rmagine::VRAM_CUDA> particle_attrs_new,
  const ParticleUpdateDynamicConfig& config)
{
  const unsigned int n_particles = particle_poses.size();
  constexpr unsigned int blockSize = 1024;
  const unsigned int gridSize = (particle_poses.size() + blockSize - 1) / blockSize;

  residual_resample_kernel<<<gridSize, blockSize>>>(
    particle_poses.raw(), particle_attrs.raw(), stats.raw(), rstates.raw(),
    particle_poses_new.raw(), particle_attrs_new.raw(), n_particles, config);
}




} // namespace rmcl